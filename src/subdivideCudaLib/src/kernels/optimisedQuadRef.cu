#include "hip/hip_runtime.h"
#include "quadRefinement.cuh"
#include "../util/util.cuh"

inline __device__ int next(int h) { return h % 4 == 3 ? h - 3 : h + 1; }

inline __device__ int prev(int h) { return h % 4 == 0 ? h + 3 : h - 1; }

inline __device__ int face(int h) { return h / 4; }

__global__ void optimisedSubdivide(DeviceMesh* in, DeviceMesh* out, int v0) {
    __shared__ float facePointsX[FACES_PER_BLOCK];
    __shared__ float facePointsY[FACES_PER_BLOCK];
    __shared__ float facePointsZ[FACES_PER_BLOCK];

    int vd = in->numVerts;
    int fd = in->numFaces;
    int ed = in->numEdges;

    int ti = threadIdx.x / 4;
    int t2 = threadIdx.x % 4;

    int start = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int h = start; h < in->numHalfEdges; h += stride) {
        // not all threads in the warp execute this, but it should eliminate the need for thread sync
        if(t2 == 0) {
            // reset shared memory
            facePointsX[ti] = 0;
            facePointsY[ti] = 0;
            facePointsZ[ti] = 0;
        }
        // edge refinement
        int hp = prev(h);
        int he = in->edges[h];
        int v = in->verts[h];
        int ht = in->twins[h];

        out->twins[4 * h] = ht < 0 ? -1 : 4 * next(ht) + 3;
        out->twins[4 * h + 1] = 4 * next(h) + 2;
        out->twins[4 * h + 2] = 4 * hp + 1;
        out->twins[4 * h + 3] = 4 * in->twins[hp];

        out->verts[4 * h] = v;
        out->verts[4 * h + 1] = vd + fd + he;
        out->verts[4 * h + 2] = vd + face(h);
        out->verts[4 * h + 3] = vd + fd + in->edges[hp];

        out->edges[4 * h] = h > ht ? 2 * he : 2 * he + 1;
        out->edges[4 * h + 1] = 2 * ed + h;
        out->edges[4 * h + 2] = 2 * ed + hp;
        out->edges[4 * h + 3] = hp > in->twins[hp] ? 2 * in->edges[hp] + 1 : 2 * in->edges[hp];
        
        // face points
        float invX = in->xCoords[v];
        float invY = in->yCoords[v];
        float invZ = in->zCoords[v];

        atomicAdd(&facePointsX[ti], invX / 4.0f);
        atomicAdd(&facePointsY[ti], invY / 4.0f);
        atomicAdd(&facePointsZ[ti], invZ / 4.0f);

        // edge points
        float x, y, z;
        // boundary

        int k = in->verts[next(h)];
        float edgex = (invX + in->xCoords[k]) / 2.0f;
        float edgey = (invY + in->yCoords[k]) / 2.0f;
        float edgez = (invZ + in->zCoords[k]) / 2.0f;
        
        if(ht < 0) {
            x = edgex;
            y = edgey;
            z = edgez;      
        } else {
            // average the vertex of this vertex and the face point
            x = (invX + facePointsX[ti]) / 4.0f;
            y = (invY + facePointsY[ti]) / 4.0f;
            z = (invZ + facePointsZ[ti]) / 4.0f;
        }    
        int j = vd + fd + he;
        atomicAdd(&out->xCoords[j], x);
        atomicAdd(&out->yCoords[j], y);
        atomicAdd(&out->zCoords[j], z);

        // this is pretty awesome trick yes
        float n = v >= v0 ? 4 : valence(h, in);
        if(ht < 0) {
            out->xCoords[v] = invX;
            out->yCoords[v] = invY;
            out->zCoords[v] = invZ;
        } else if (n >= 0) {
            float n2 = n * n;
            x = (2 * edgex + facePointsX[ti] + (n - 3) * invX) / n2;
            y = (2 * edgey + facePointsY[ti] + (n - 3) * invY) / n2;
            z = (2 * edgez + facePointsZ[ti] + (n - 3) * invZ) / n2;
            atomicAdd(&out->xCoords[v], x);
            atomicAdd(&out->yCoords[v], y);
            atomicAdd(&out->zCoords[v], z);
        }
        
        if(t2 == 0) {
            int ind = vd + face(h);
            out->xCoords[ind] = facePointsX[ti];
            out->yCoords[ind] = facePointsY[ti];
            out->zCoords[ind] = facePointsZ[ti];
        }
    }  
}
