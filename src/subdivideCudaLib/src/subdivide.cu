#include "hip/hip_runtime.h"
#include "../libheaders/subdivide.h"

#include <stdlib.h>
#include <stdio.h>
#include "math.h"

#include "mesh.cuh"
#include "util/util.cuh"
#include "kernelInvoker.cuh"
#include "deviceCommunication.cuh"


void subdivide(Mesh* mesh, int subdivisionLevel) {
	printf("Starting Subdvision\n");
	
	// use double buffering; calculate final number of half edges and numVerts and allocat out and in arrays
	// switch each subdivision level
	Mesh in = makeEmptyCopy(mesh);
	Mesh out = makeEmptyCopy(mesh);

	int finalNumberOfHalfEdges = pow(4, subdivisionLevel) * mesh->numHalfEdges;
	// assumes quad mesh
	int v1 = mesh->numVerts + mesh->numFaces + mesh->numEdges;
	int e1 = 2 * mesh->numEdges + mesh->numHalfEdges;
	int f1 = mesh->numHalfEdges;
	int finalNumberOfVerts = v1 + pow(2, subdivisionLevel - 1) * (e1 + (pow(2, subdivisionLevel) -1) * f1);

	allocateDeviceMemory(&in, finalNumberOfVerts, finalNumberOfHalfEdges);
	allocateDeviceMemory(&out, finalNumberOfVerts, finalNumberOfHalfEdges);

	hipDeviceSynchronize();

	copyHostToDeviceMesh(mesh, &in);
	
	hipDeviceSynchronize();

	performSubdivision(in, out, subdivisionLevel);
	// device is synced after this call
	// result is in out
	reallocHostMemory(mesh, &out);	
	copyHostToDeviceMesh(mesh, &out);
	
	hipDeviceSynchronize();

	freeDeviceMesh(&in);
	freeDeviceMesh(&out);

	printf("Subdivision Complete!\n");
}

// returns the number of milsecs the subdivision took
double timedSubdivision(float* xCoords, float* yCoords, float* zCoords, int numVerts, int numHalfEdges, int numFaces, int numEdges, int* twins, int* nexts, int* prevs, int* verts, int* edges, int* faces, int subdivisionLevel) {
	printf("Setting up mesh\n");
	Mesh baseMesh = initMesh(numVerts, numHalfEdges, numFaces, numEdges);

	baseMesh.xCoords = xCoords;
	baseMesh.yCoords = yCoords;
	baseMesh.zCoords = zCoords;

	baseMesh.twins = twins;
	baseMesh.nexts = nexts;
	baseMesh.prevs = prevs;
	baseMesh.verts = verts;
	baseMesh.edges = edges;
	baseMesh.faces = faces;

	toObjFile(&baseMesh);

	subdivide(&baseMesh, subdivisionLevel);

	return 0.0;
}
