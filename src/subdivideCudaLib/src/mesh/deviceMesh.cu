#include <stdlib.h>
#include <stdio.h>

#include "devicemesh.cuh"
#include "../util/util.cuh"

DeviceMesh createEmptyCopyOnDevice(Mesh* mesh) {
    return initEmptyDeviceMesh(mesh->numVerts, mesh->numHalfEdges, mesh->numFaces, mesh->numEdges);
}

void setDevicePointerValue(int** loc, int val) {
    hipError_t cuda_ret;
    cuda_ret = hipMalloc((void**)loc, sizeof(int));
    cudaErrCheck(cuda_ret, "Unable to allocate device int pointer val");
    cuda_ret = hipMemcpy(*loc, &val, sizeof(int), hipMemcpyHostToDevice);
    cudaErrCheck(cuda_ret, "Unable to copy val to device pointer");
}

DeviceMesh initEmptyDeviceMesh(int numVerts, int numHalfEdges, int numFaces, int numEdges) {
	DeviceMesh mesh = {};
    mesh.numVerts = numVerts;
    mesh.numHalfEdges = numHalfEdges;
    mesh.numFaces = numFaces;
    mesh.numEdges = numEdges;

    printf("\n--\nCopying %d %d %d %d\n --\n", numVerts, numHalfEdges, numFaces, numEdges);
    // setDevicePointerValue(&mesh.numVerts, numVerts);
    // setDevicePointerValue(&mesh.numHalfEdges, numHalfEdges);
    // setDevicePointerValue(&mesh.numFaces, numFaces);
    // setDevicePointerValue(&mesh.numEdges, numEdges);
	return mesh;
} 

DeviceMesh* toDevicePointer(DeviceMesh* mesh_h) {
    hipError_t cuda_ret;
    DeviceMesh* mesh_d;
    cuda_ret = hipMalloc((void**)&mesh_d, sizeof(DeviceMesh));
    cudaErrCheck(cuda_ret, "Unable to allocate device struct val");
    cuda_ret = hipMemcpy(mesh_d, mesh_h, sizeof(DeviceMesh), hipMemcpyHostToDevice);
    cudaErrCheck(cuda_ret, "Unable to copy struct to device pointer");
	return mesh_d;
} 

DeviceMesh devicePointerToHostMesh(DeviceMesh* mesh_d) {
    hipError_t cuda_ret;
    DeviceMesh mesh_h = {};
    cuda_ret = hipMemcpy(&mesh_h, mesh_d, sizeof(DeviceMesh), hipMemcpyDeviceToHost);
    cudaErrCheck(cuda_ret, "Unable to copy struct to host pointer");
	return mesh_h;
} 

void freeDeviceMesh(DeviceMesh* mesh) {
    // Mesh is device pointer
    hipFree(mesh->xCoords);
    hipFree(mesh->yCoords);
    hipFree(mesh->zCoords);
    hipFree(mesh->twins);
    hipFree(mesh->nexts);
    hipFree(mesh->prevs);
    hipFree(mesh->verts);
    hipFree(mesh->edges);
    hipFree(mesh->faces);
}
