#include <stdlib.h>
#include <stdio.h>

#include "devicemesh.cuh"
#include "../util/util.cuh"

DeviceMesh createEmptyCopyOnDevice(Mesh* mesh) {
    return initEmptyDeviceMesh(mesh->numVerts, mesh->numHalfEdges, mesh->numFaces, mesh->numEdges);
}

void setDevicePointerValue(int** loc, int val) {
    hipError_t cuda_ret;
    cuda_ret = hipMalloc((void**)loc, sizeof(int));
    cudaErrCheck(cuda_ret, "Unable to allocate device int pointer val");
    cuda_ret = hipMemcpy(*loc, &val, sizeof(int), hipMemcpyHostToDevice);
    cudaErrCheck(cuda_ret, "Unable to copy val to device pointer");
}

DeviceMesh initEmptyDeviceMesh(int numVerts, int numHalfEdges, int numFaces, int numEdges) {
	DeviceMesh mesh = {};
    printf("\n--\nCopying %d %d %d %d\n --\n", numVerts, numHalfEdges, numFaces, numEdges);
    setDevicePointerValue(&mesh.numVerts, numVerts);
    setDevicePointerValue(&mesh.numHalfEdges, numHalfEdges);
    setDevicePointerValue(&mesh.numFaces, numFaces);
    setDevicePointerValue(&mesh.numEdges, numEdges);
	return mesh;
} 

void freeDeviceMesh(DeviceMesh* mesh) {
    hipFree(mesh->xCoords);
    hipFree(mesh->yCoords);
    hipFree(mesh->zCoords);
    hipFree(mesh->twins);
    hipFree(mesh->nexts);
    hipFree(mesh->prevs);
    hipFree(mesh->verts);
    hipFree(mesh->edges);
    hipFree(mesh->faces);
}
