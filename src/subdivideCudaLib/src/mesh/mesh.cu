#include <stdlib.h>
#include <stdio.h>

#include "mesh.cuh"

Mesh makeEmptyCopy(Mesh* mesh) {
    return initMesh(mesh->numVerts, mesh->numHalfEdges, mesh->numFaces, mesh->numEdges);
}

Mesh initMesh(int numVerts, int numHalfEdges, int numFaces, int numEdges) {
	Mesh mesh;
	mesh.numVerts = numVerts;
	mesh.numHalfEdges = numHalfEdges;
	mesh.numEdges = numEdges;
	mesh.numFaces = numFaces;
    mesh.nexts = NULL;
    mesh.prevs = NULL;
    mesh.faces = NULL;
    mesh.twins = NULL;
    mesh.edges = NULL;
    mesh.verts = NULL;
	return mesh;
} 

void freeDeviceMesh(Mesh* mesh) {
    hipFree(mesh->xCoords);
    hipFree(mesh->yCoords);
    hipFree(mesh->zCoords);
    hipFree(mesh->twins);
    hipFree(mesh->nexts);
    hipFree(mesh->prevs);
    hipFree(mesh->verts);
    hipFree(mesh->edges);
    hipFree(mesh->faces);
}

void freeMesh(Mesh* mesh) {
    free(mesh->xCoords);
    free(mesh->yCoords);
    free(mesh->zCoords);
    free(mesh->twins);
    free(mesh->nexts);
    free(mesh->prevs);
    free(mesh->verts);
    free(mesh->edges);
    free(mesh->faces);
}

void toObjFile(Mesh* mesh) {
    printf("Writing mesh to file..\n");
    // TODO: add name of object file
    FILE *objFile = fopen("result.obj", "w");
    if (objFile == NULL)
    {
        printf("Error opening or creating .obj file!\n");
        exit(1);
    }
    // print vertices
    for(int v = 0; v < mesh->numVerts; v++) {
        fprintf(objFile, "v %.6lf %.6lf %.6lf\n", mesh->xCoords[v], mesh->yCoords[v], mesh->zCoords[v]);
    }
    fprintf(objFile, "# Numfaces: %d\n\n", mesh->numFaces); 
    // list of face indices
    for(int f = 0; f < mesh->numFaces; f++) {
        fprintf(objFile, "f");
        for(int v = 0; v < 4; v++) {
            // indices in .obj start at 1
            fprintf(objFile, " %d", mesh->verts[f*4 + v] + 1);
        }
        fprintf(objFile, "\n");
    }
    fclose(objFile);
}
