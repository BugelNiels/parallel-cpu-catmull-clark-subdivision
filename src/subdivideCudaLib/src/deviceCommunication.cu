#include <stdlib.h>
#include <stdio.h>

#include "deviceCommunication.cuh"
#include "util/util.cuh"


// m = number of vertices in vD; n = number of half edges in vD
void allocateDeviceMemory(DeviceMesh* deviceMesh, int m, int n, int n0, int isQuad) {
	hipError_t cuda_ret;
	Timer timer;
	printf("Allocating device variables...\n"); fflush(stdout);
    startTime(&timer);
    printf("    Allocating size %d: %d bytes\n", m, m * sizeof(float));
    cuda_ret = hipMalloc((void**)&deviceMesh->xCoords, m * sizeof(float));
    cudaErrCheck(cuda_ret, "Unable to allocate device memory for X coordinates");
    cuda_ret = hipMalloc((void**)&deviceMesh->yCoords, m * sizeof(float));
    cudaErrCheck(cuda_ret, "Unable to allocate device memory for Y coordinates");
	cuda_ret = hipMalloc((void**)&deviceMesh->zCoords, m * sizeof(float));
    cudaErrCheck(cuda_ret, "Unable to allocate device memory for Z coordinates");


    printf("    Allocating size %d: %d bytes\n", n, n * sizeof(int));
	cuda_ret = hipMalloc((void**)&deviceMesh->twins, n * sizeof(int));
    cudaErrCheck(cuda_ret, "Unable to allocate device memory for twin array");
	cuda_ret = hipMalloc((void**)&deviceMesh->verts, n * sizeof(int));
    cudaErrCheck(cuda_ret, "Unable to allocate device memory for vert array");
	cuda_ret = hipMalloc((void**)&deviceMesh->edges, n * sizeof(int));
    cudaErrCheck(cuda_ret, "Unable to allocate device memory for edge array");

    if(isQuad == 0) {
        //only allocate enough for the very first mesh
        cuda_ret = hipMalloc((void**)&deviceMesh->nexts, n0 * sizeof(int));
        cudaErrCheck(cuda_ret, "Unable to allocate device memory for next array");
        cuda_ret = hipMalloc((void**)&deviceMesh->prevs, n0 * sizeof(int));
        cudaErrCheck(cuda_ret, "Unable to allocate device memory for prev array");
        cuda_ret = hipMalloc((void**)&deviceMesh->faces, n0 * sizeof(int));
        cudaErrCheck(cuda_ret, "Unable to allocate device memory for face array");
    }

    stopTime(&timer); printf("Allocation took: %f s\n\n", elapsedTime(timer));
}

int getDeviceVal(int** deviceLoc) {
	hipError_t cuda_ret;
    int val = 0;
    cuda_ret = hipMemcpy(&val, *deviceLoc, sizeof(int), hipMemcpyDeviceToHost);
    cudaErrCheck(cuda_ret, "Unable to copy val to device pointer");
    return val;
}

void reallocHostMemory(Mesh* hostMesh, DeviceMesh* deviceMesh) {

    
    hostMesh->numEdges = getDeviceVal(&deviceMesh->numEdges);
    hostMesh->numFaces = getDeviceVal(&deviceMesh->numFaces);
    hostMesh->numHalfEdges = getDeviceVal(&deviceMesh->numHalfEdges);
    hostMesh->numVerts = getDeviceVal(&deviceMesh->numVerts);

    hostMesh->xCoords = (float*)realloc(hostMesh->xCoords, hostMesh->numVerts);
    hostMesh->yCoords = (float*)realloc(hostMesh->yCoords, hostMesh->numVerts);
    hostMesh->zCoords = (float*)realloc(hostMesh->zCoords, hostMesh->numVerts);

    hostMesh->twins = (int*)realloc(hostMesh->twins, hostMesh->numHalfEdges);
    hostMesh->verts = (int*)realloc(hostMesh->verts, hostMesh->numHalfEdges);
    hostMesh->edges = (int*)realloc(hostMesh->edges, hostMesh->numHalfEdges);
}

void copyHostToDeviceMesh(Mesh* from, DeviceMesh* to, int isQuad) {
	Timer timer;

	printf("Copying mesh from host to device...\n"); fflush(stdout);
    startTime(&timer);

    hipError_t cuda_ret;

    int m = from->numVerts;
    if(m == 0) {
        printf("Source mesh coords are empty"); 
        return;
    }
    printf("    Copying %d: %d bytes\n", m, m * sizeof(float));
	cuda_ret = hipMemcpy(to->xCoords, from->xCoords, m * sizeof(float), hipMemcpyHostToDevice);
    cudaErrCheck(cuda_ret, "Unable to copy x-coordinates to the device");
	cuda_ret = hipMemcpy(to->yCoords, from->yCoords, m * sizeof(float), hipMemcpyHostToDevice);
    cudaErrCheck(cuda_ret, "Unable to copy y-coordinates to the device")
	cuda_ret = hipMemcpy(to->zCoords, from->zCoords, m * sizeof(float), hipMemcpyHostToDevice);
    cudaErrCheck(cuda_ret, "Unable to copy z-coordinates to the device");

	int n = from->numHalfEdges;
    if(n == 0) {
        printf("Source mesh properties are empty"); 
        return;
    }
    printf("    Copying %d: %d bytes\n", n, n * sizeof(float));
	cuda_ret = hipMemcpy(to->twins, from->twins, n * sizeof(int), hipMemcpyHostToDevice);
    cudaErrCheck(cuda_ret, "Unable to copy twins to the device");
	cuda_ret = hipMemcpy(to->verts, from->verts, n * sizeof(int), hipMemcpyHostToDevice);
    cudaErrCheck(cuda_ret, "Unable to copy verts to the device");
	cuda_ret = hipMemcpy(to->edges, from->edges, n * sizeof(int), hipMemcpyHostToDevice);
    cudaErrCheck(cuda_ret, "Unable to copy edges to the device");
    
    if(isQuad == 0) {
        cuda_ret = hipMemcpy(to->nexts, from->nexts, n * sizeof(int), hipMemcpyHostToDevice);
        cudaErrCheck(cuda_ret, "Unable to copy nexts to the device");
        cuda_ret = hipMemcpy(to->prevs, from->prevs, n * sizeof(int), hipMemcpyHostToDevice);
        cudaErrCheck(cuda_ret, "Unable to copy prevs to the device"); 
        cuda_ret = hipMemcpy(to->faces, from->faces, n * sizeof(int), hipMemcpyHostToDevice);
        cudaErrCheck(cuda_ret, "Unable to copy faces to the device");
    }

	stopTime(&timer); printf("Copy to device took %f s\n\n", elapsedTime(timer));
}

void copyDeviceMeshToHostMesh(Mesh* to, DeviceMesh* from) {
	Timer timer;

	printf("Copying mesh from device back to host...\n"); fflush(stdout);
    startTime(&timer);

    hipError_t cuda_ret;

    // to already has the correct values for num..
    int m = to->numVerts;
    if(m == 0) {
        printf("Source mesh coords are empty"); 
        return;
    }
    printf("    Copying %d: %d bytes\n", m, m * sizeof(float));
	cuda_ret = hipMemcpy(to->xCoords, from->xCoords, m * sizeof(float), hipMemcpyDeviceToHost);
    cudaErrCheck(cuda_ret, "Unable to copy x-coordinates from the device");
	cuda_ret = hipMemcpy(to->yCoords, from->yCoords, m * sizeof(float), hipMemcpyDeviceToHost);
    cudaErrCheck(cuda_ret, "Unable to copy y-coordinates from the device")
	cuda_ret = hipMemcpy(to->zCoords, from->zCoords, m * sizeof(float), hipMemcpyDeviceToHost);
    cudaErrCheck(cuda_ret, "Unable to copy z-coordinates from the device");

	int n = to->numHalfEdges;
    if(n == 0) {
        printf("Source mesh properties are empty"); 
        return;
    }
    printf("    Copying %d: %d bytes\n", n, n * sizeof(float));
	cuda_ret = hipMemcpy(to->twins, from->twins, n * sizeof(int), hipMemcpyDeviceToHost);
    cudaErrCheck(cuda_ret, "Unable to copy twins from the device");
	cuda_ret = hipMemcpy(to->verts, from->verts, n * sizeof(int), hipMemcpyDeviceToHost);
    cudaErrCheck(cuda_ret, "Unable to copy verts from the device");
	cuda_ret = hipMemcpy(to->edges, from->edges, n * sizeof(int), hipMemcpyDeviceToHost);
    cudaErrCheck(cuda_ret, "Unable to copy edges from the device");
 

	stopTime(&timer); printf("Copy to host took: %f s\n\n", elapsedTime(timer));
}